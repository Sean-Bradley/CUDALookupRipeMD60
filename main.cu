#include "hip/hip_runtime.h"
#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <ctime>

using namespace std;

extern "C" __global__ static void kernel(unsigned char* output, int outputLength, unsigned char* query, int queryLength, int N);
__device__ static unsigned char* RIPEMD160Compute(unsigned char* output, int outputLength, unsigned char* query, int queryLength, int idx, int idy, int idz);
__device__ static int RL(int x, int n);
__device__ static int F1(int x, int y, int z);
__device__ static int F2(int x, int y, int z);
__device__ static int F3(int x, int y, int z);
__device__ static int F4(int x, int y, int z);
__device__ static int F5(int x, int y, int z);
__device__ static int* ProcessBlock(int* X, int XLen0, int* H, int HLength);
extern "C" __global__ static void kernel(unsigned char* output, int outputLength, unsigned char* query, int queryLength, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		for (int v1 = 0x73; v1 < 0x74; v1++)
		{
			RIPEMD160Compute(output, outputLength, query, queryLength, idx & 0x000000FF, idx >> 8, v1);
		}
	}
}

__device__ static unsigned char* RIPEMD160Compute(unsigned char* output, int outputLength, unsigned char* query, int queryLength, int idx, int idy, int idz)
{
	int InputLength = 8;//32;
	int InputLengthAtStart = InputLength;
	unsigned char * input = new unsigned char[InputLength];

	//password = 70 61 73 73 77 6f 72 64
	input[0] = idx;
	input[1] = idy;
	input[2] = idz;
	input[3] = 0x73;
	input[4] = 0x77;
	input[5] = 0x6f;
	input[6] = 0x72;
	input[7] = 0x64;

	int * H = new int[5];
	int HLength = 5;

	H[0] = 1732584193;
	H[1] = -271733879;
	H[2] = -1732584194;
	H[3] = 271733878;
	H[4] = -1009589776;

	int xOffset = 0;
	int byteCount = 0;
	int xBufOffset = 0;
	int offset = 0;

	int XLength = 16;
	int* X = new int[16];
	for (int i = 0; i < XLength; i++) X[i] = 0;

	unsigned char * xBuf = new unsigned char[4];
	xBuf[0] = 0;
	xBuf[1] = 0;
	xBuf[2] = 0;
	xBuf[3] = 0;
	int xBufLength2use = 4;

	while (xBufOffset != 0 && InputLength > 0)
	{
		xBuf[xBufOffset++] = input[offset];
		if (xBufOffset == xBufLength2use)
		{
			X[xOffset++] = (int)((xBuf[0] & 255) | (int)(xBuf[1] & 255) << 8 | (int)(xBuf[2] & 255) << 16 | (int)(xBuf[3] & 255) << 24);
			if (xOffset == 16)
			{
				H = ProcessBlock(X, XLength, H, HLength);
				xOffset = 0;
			}
			xBufOffset = 0;
		}
		byteCount++;
		offset++;
		InputLength--;
	}

	while (InputLength > xBufLength2use)
	{
		X[xOffset++] = (int)((input[offset] & 255) | (int)(input[offset + 1] & 255) << 8 | (int)(input[offset + 2] & 255) << 16 | (int)(input[offset + 3] & 255) << 24);
		if (xOffset == 16)
		{
			H = ProcessBlock(X, XLength, H, HLength);
			xOffset = 0;
		}
		offset += xBufLength2use;
		InputLength -= xBufLength2use;
		byteCount += xBufLength2use;
	}
	while (InputLength > 0)
	{
		xBuf[xBufOffset++] = input[offset];
		if (xBufOffset == xBufLength2use)
		{
			X[xOffset++] = (int)((xBuf[0] & 255) | (int)(xBuf[1] & 255) << 8 | (int)(xBuf[2] & 255) << 16 | (int)(xBuf[3] & 255) << 24);
			if (xOffset == 16)
			{
				H = ProcessBlock(X, XLength, H, HLength);
				xOffset = 0;
			}
			xBufOffset = 0;
		}
		byteCount++;
		offset++;
		InputLength--;
	}
	long long num6 = (long long)(byteCount << 3);
	xBuf[xBufOffset++] = 128;
	if (xBufOffset == xBufLength2use)
	{
		X[xOffset++] = (int)((xBuf[0] & 255) | (int)(xBuf[1] & 255) << 8 | (int)(xBuf[2] & 255) << 16 | (int)(xBuf[3] & 255) << 24);
		if (xOffset == 16)
		{
			H = ProcessBlock(X, XLength, H, HLength);
			xOffset = 0;
		}
		xBufOffset = 0;
	}
	byteCount++;
	while (xBufOffset != 0)
	{
		xBuf[xBufOffset++] = 0;
		if (xBufOffset == xBufLength2use)
		{
			X[xOffset++] = (int)((xBuf[0] & 255) | (int)(xBuf[1] & 255) << 8 | (int)(xBuf[2] & 255) << 16 | (int)(xBuf[3] & 255) << 24);
			xBufOffset = 0;
		}
		byteCount++;
	}
	if (xOffset > 14)
	{
		H = ProcessBlock(X, XLength, H, HLength);
		xOffset = 0;
	}

	X[14] = (int)(num6 & (long long)((unsigned long long) - 1));
	X[15] = (int)((unsigned int)num6 >> 32);

	H = ProcessBlock(X, XLength, H, HLength);
	xOffset = 0;

	unsigned char * result = new unsigned char[20];
	result[0] = (unsigned char)H[0];
	result[1] = (unsigned char)((unsigned int)H[0] >> 8);
	result[2] = (unsigned char)((unsigned int)H[0] >> 16);
	result[3] = (unsigned char)((unsigned int)H[0] >> 24);
	result[4] = (unsigned char)H[1];
	result[5] = (unsigned char)((unsigned int)H[1] >> 8);
	result[6] = (unsigned char)((unsigned int)H[1] >> 16);
	result[7] = (unsigned char)((unsigned int)H[1] >> 24);
	result[8] = (unsigned char)H[2];
	result[9] = (unsigned char)((unsigned int)H[2] >> 8);
	result[10] = (unsigned char)((unsigned int)H[2] >> 16);
	result[11] = (unsigned char)((unsigned int)H[2] >> 24);
	result[12] = (unsigned char)H[3];
	result[13] = (unsigned char)((unsigned int)H[3] >> 8);
	result[14] = (unsigned char)((unsigned int)H[3] >> 16);
	result[15] = (unsigned char)((unsigned int)H[3] >> 24);
	result[16] = (unsigned char)H[4];
	result[17] = (unsigned char)((unsigned int)H[4] >> 8);
	result[18] = (unsigned char)((unsigned int)H[4] >> 16);
	result[19] = (unsigned char)((unsigned int)H[4] >> 24);

	bool cont = true;
	int t = 0;
	while (cont)
	{
		if (result[t] == query[t])
		{
			cont = true;
			if (t == 19)
			{
				//full match found
				printf("%d : *** match found ***n", idx);
				for (int i = 0; i < 20; i++)
				{
					printf("%02X", result[i]);
					if (i < InputLengthAtStart)
					{
						output[i] = input[i];
					}
					outputLength = InputLengthAtStart;
				}
				printf("n");
				break;
			}
		}
		else
		{
			cont = false;
			break;
		}
		t++;
	}

	free(input);
	free(H);
	free(xBuf);
	free(result);
	free(output);
	free(query);

	return output;
}

__device__ static int RL(int x, int n)
{
	return x << (n & 31) | (unsigned int)x >> (32 - n & 31);
}

__device__ static int F1(int x, int y, int z)
{
	return x ^ y ^ z;
}

__device__ static int F2(int x, int y, int z)
{
	return (x & y) | (~x & z);
}

__device__ static int F3(int x, int y, int z)
{
	return (x | ~y) ^ z;
}

__device__ static int F4(int x, int y, int z)
{
	return (x & z) | (y & ~z);
}

__device__ static int F5(int x, int y, int z)
{
	return x ^ (y | ~z);
}

__device__ static int* ProcessBlock(int* X, int XLen0, int* H, int HLength)
{
	int a = H[0];
	int aa = H[0];
	int b = H[1];
	int bb = H[1];
	int c = H[2];
	int cc = H[2];
	int d = H[3];
	int dd = H[3];
	int e = H[4];
	int ee = H[4];
	a = RL(a + F1(b, c, d) + X[0], 11) + e; c = RL(c, 10);
	e = RL(e + F1(a, b, c) + X[1], 14) + d; b = RL(b, 10);
	d = RL(d + F1(e, a, b) + X[2], 15) + c; a = RL(a, 10);
	c = RL(c + F1(d, e, a) + X[3], 12) + b; e = RL(e, 10);
	b = RL(b + F1(c, d, e) + X[4], 5) + a; d = RL(d, 10);
	a = RL(a + F1(b, c, d) + X[5], 8) + e; c = RL(c, 10);
	e = RL(e + F1(a, b, c) + X[6], 7) + d; b = RL(b, 10);
	d = RL(d + F1(e, a, b) + X[7], 9) + c; a = RL(a, 10);
	c = RL(c + F1(d, e, a) + X[8], 11) + b; e = RL(e, 10);
	b = RL(b + F1(c, d, e) + X[9], 13) + a; d = RL(d, 10);
	a = RL(a + F1(b, c, d) + X[10], 14) + e; c = RL(c, 10);
	e = RL(e + F1(a, b, c) + X[11], 15) + d; b = RL(b, 10);
	d = RL(d + F1(e, a, b) + X[12], 6) + c; a = RL(a, 10);
	c = RL(c + F1(d, e, a) + X[13], 7) + b; e = RL(e, 10);
	b = RL(b + F1(c, d, e) + X[14], 9) + a; d = RL(d, 10);
	a = RL(a + F1(b, c, d) + X[15], 8) + e; c = RL(c, 10);
	aa = RL(aa + F5(bb, cc, dd) + X[5] + 1352829926, 8) + ee; cc = RL(cc, 10);
	ee = RL(ee + F5(aa, bb, cc) + X[14] + 1352829926, 9) + dd; bb = RL(bb, 10);
	dd = RL(dd + F5(ee, aa, bb) + X[7] + 1352829926, 9) + cc; aa = RL(aa, 10);
	cc = RL(cc + F5(dd, ee, aa) + X[0] + 1352829926, 11) + bb; ee = RL(ee, 10);
	bb = RL(bb + F5(cc, dd, ee) + X[9] + 1352829926, 13) + aa; dd = RL(dd, 10);
	aa = RL(aa + F5(bb, cc, dd) + X[2] + 1352829926, 15) + ee; cc = RL(cc, 10);
	ee = RL(ee + F5(aa, bb, cc) + X[11] + 1352829926, 15) + dd; bb = RL(bb, 10);
	dd = RL(dd + F5(ee, aa, bb) + X[4] + 1352829926, 5) + cc; aa = RL(aa, 10);
	cc = RL(cc + F5(dd, ee, aa) + X[13] + 1352829926, 7) + bb; ee = RL(ee, 10);
	bb = RL(bb + F5(cc, dd, ee) + X[6] + 1352829926, 7) + aa; dd = RL(dd, 10);
	aa = RL(aa + F5(bb, cc, dd) + X[15] + 1352829926, 8) + ee; cc = RL(cc, 10);
	ee = RL(ee + F5(aa, bb, cc) + X[8] + 1352829926, 11) + dd; bb = RL(bb, 10);
	dd = RL(dd + F5(ee, aa, bb) + X[1] + 1352829926, 14) + cc; aa = RL(aa, 10);
	cc = RL(cc + F5(dd, ee, aa) + X[10] + 1352829926, 14) + bb; ee = RL(ee, 10);
	bb = RL(bb + F5(cc, dd, ee) + X[3] + 1352829926, 12) + aa; dd = RL(dd, 10);
	aa = RL(aa + F5(bb, cc, dd) + X[12] + 1352829926, 6) + ee; cc = RL(cc, 10);
	e = RL(e + F2(a, b, c) + X[7] + 1518500249, 7) + d; b = RL(b, 10);
	d = RL(d + F2(e, a, b) + X[4] + 1518500249, 6) + c; a = RL(a, 10);
	c = RL(c + F2(d, e, a) + X[13] + 1518500249, 8) + b; e = RL(e, 10);
	b = RL(b + F2(c, d, e) + X[1] + 1518500249, 13) + a; d = RL(d, 10);
	a = RL(a + F2(b, c, d) + X[10] + 1518500249, 11) + e; c = RL(c, 10);
	e = RL(e + F2(a, b, c) + X[6] + 1518500249, 9) + d; b = RL(b, 10);
	d = RL(d + F2(e, a, b) + X[15] + 1518500249, 7) + c; a = RL(a, 10);
	c = RL(c + F2(d, e, a) + X[3] + 1518500249, 15) + b; e = RL(e, 10);
	b = RL(b + F2(c, d, e) + X[12] + 1518500249, 7) + a; d = RL(d, 10);
	a = RL(a + F2(b, c, d) + X[0] + 1518500249, 12) + e; c = RL(c, 10);
	e = RL(e + F2(a, b, c) + X[9] + 1518500249, 15) + d; b = RL(b, 10);
	d = RL(d + F2(e, a, b) + X[5] + 1518500249, 9) + c; a = RL(a, 10);
	c = RL(c + F2(d, e, a) + X[2] + 1518500249, 11) + b; e = RL(e, 10);
	b = RL(b + F2(c, d, e) + X[14] + 1518500249, 7) + a; d = RL(d, 10);
	a = RL(a + F2(b, c, d) + X[11] + 1518500249, 13) + e; c = RL(c, 10);
	e = RL(e + F2(a, b, c) + X[8] + 1518500249, 12) + d; b = RL(b, 10);
	ee = RL(ee + F4(aa, bb, cc) + X[6] + 1548603684, 9) + dd; bb = RL(bb, 10);
	dd = RL(dd + F4(ee, aa, bb) + X[11] + 1548603684, 13) + cc; aa = RL(aa, 10);
	cc = RL(cc + F4(dd, ee, aa) + X[3] + 1548603684, 15) + bb; ee = RL(ee, 10);
	bb = RL(bb + F4(cc, dd, ee) + X[7] + 1548603684, 7) + aa; dd = RL(dd, 10);
	aa = RL(aa + F4(bb, cc, dd) + X[0] + 1548603684, 12) + ee; cc = RL(cc, 10);
	ee = RL(ee + F4(aa, bb, cc) + X[13] + 1548603684, 8) + dd; bb = RL(bb, 10);
	dd = RL(dd + F4(ee, aa, bb) + X[5] + 1548603684, 9) + cc; aa = RL(aa, 10);
	cc = RL(cc + F4(dd, ee, aa) + X[10] + 1548603684, 11) + bb; ee = RL(ee, 10);
	bb = RL(bb + F4(cc, dd, ee) + X[14] + 1548603684, 7) + aa; dd = RL(dd, 10);
	aa = RL(aa + F4(bb, cc, dd) + X[15] + 1548603684, 7) + ee; cc = RL(cc, 10);
	ee = RL(ee + F4(aa, bb, cc) + X[8] + 1548603684, 12) + dd; bb = RL(bb, 10);
	dd = RL(dd + F4(ee, aa, bb) + X[12] + 1548603684, 7) + cc; aa = RL(aa, 10);
	cc = RL(cc + F4(dd, ee, aa) + X[4] + 1548603684, 6) + bb; ee = RL(ee, 10);
	bb = RL(bb + F4(cc, dd, ee) + X[9] + 1548603684, 15) + aa; dd = RL(dd, 10);
	aa = RL(aa + F4(bb, cc, dd) + X[1] + 1548603684, 13) + ee; cc = RL(cc, 10);
	ee = RL(ee + F4(aa, bb, cc) + X[2] + 1548603684, 11) + dd; bb = RL(bb, 10);
	d = RL(d + F3(e, a, b) + X[3] + 1859775393, 11) + c; a = RL(a, 10);
	c = RL(c + F3(d, e, a) + X[10] + 1859775393, 13) + b; e = RL(e, 10);
	b = RL(b + F3(c, d, e) + X[14] + 1859775393, 6) + a; d = RL(d, 10);
	a = RL(a + F3(b, c, d) + X[4] + 1859775393, 7) + e; c = RL(c, 10);
	e = RL(e + F3(a, b, c) + X[9] + 1859775393, 14) + d; b = RL(b, 10);
	d = RL(d + F3(e, a, b) + X[15] + 1859775393, 9) + c; a = RL(a, 10);
	c = RL(c + F3(d, e, a) + X[8] + 1859775393, 13) + b; e = RL(e, 10);
	b = RL(b + F3(c, d, e) + X[1] + 1859775393, 15) + a; d = RL(d, 10);
	a = RL(a + F3(b, c, d) + X[2] + 1859775393, 14) + e; c = RL(c, 10);
	e = RL(e + F3(a, b, c) + X[7] + 1859775393, 8) + d; b = RL(b, 10);
	d = RL(d + F3(e, a, b) + X[0] + 1859775393, 13) + c; a = RL(a, 10);
	c = RL(c + F3(d, e, a) + X[6] + 1859775393, 6) + b; e = RL(e, 10);
	b = RL(b + F3(c, d, e) + X[13] + 1859775393, 5) + a; d = RL(d, 10);
	a = RL(a + F3(b, c, d) + X[11] + 1859775393, 12) + e; c = RL(c, 10);
	e = RL(e + F3(a, b, c) + X[5] + 1859775393, 7) + d; b = RL(b, 10);
	d = RL(d + F3(e, a, b) + X[12] + 1859775393, 5) + c; a = RL(a, 10);
	dd = RL(dd + F3(ee, aa, bb) + X[15] + 1836072691, 9) + cc; aa = RL(aa, 10);
	cc = RL(cc + F3(dd, ee, aa) + X[5] + 1836072691, 7) + bb; ee = RL(ee, 10);
	bb = RL(bb + F3(cc, dd, ee) + X[1] + 1836072691, 15) + aa; dd = RL(dd, 10);
	aa = RL(aa + F3(bb, cc, dd) + X[3] + 1836072691, 11) + ee; cc = RL(cc, 10);
	ee = RL(ee + F3(aa, bb, cc) + X[7] + 1836072691, 8) + dd; bb = RL(bb, 10);
	dd = RL(dd + F3(ee, aa, bb) + X[14] + 1836072691, 6) + cc; aa = RL(aa, 10);
	cc = RL(cc + F3(dd, ee, aa) + X[6] + 1836072691, 6) + bb; ee = RL(ee, 10);
	bb = RL(bb + F3(cc, dd, ee) + X[9] + 1836072691, 14) + aa; dd = RL(dd, 10);
	aa = RL(aa + F3(bb, cc, dd) + X[11] + 1836072691, 12) + ee; cc = RL(cc, 10);
	ee = RL(ee + F3(aa, bb, cc) + X[8] + 1836072691, 13) + dd; bb = RL(bb, 10);
	dd = RL(dd + F3(ee, aa, bb) + X[12] + 1836072691, 5) + cc; aa = RL(aa, 10);
	cc = RL(cc + F3(dd, ee, aa) + X[2] + 1836072691, 14) + bb; ee = RL(ee, 10);
	bb = RL(bb + F3(cc, dd, ee) + X[10] + 1836072691, 13) + aa; dd = RL(dd, 10);
	aa = RL(aa + F3(bb, cc, dd) + X[0] + 1836072691, 13) + ee; cc = RL(cc, 10);
	ee = RL(ee + F3(aa, bb, cc) + X[4] + 1836072691, 7) + dd; bb = RL(bb, 10);
	dd = RL(dd + F3(ee, aa, bb) + X[13] + 1836072691, 5) + cc; aa = RL(aa, 10);
	c = RL(c + F4(d, e, a) + X[1] + -1894007588, 11) + b; e = RL(e, 10);
	b = RL(b + F4(c, d, e) + X[9] + -1894007588, 12) + a; d = RL(d, 10);
	a = RL(a + F4(b, c, d) + X[11] + -1894007588, 14) + e; c = RL(c, 10);
	e = RL(e + F4(a, b, c) + X[10] + -1894007588, 15) + d; b = RL(b, 10);
	d = RL(d + F4(e, a, b) + X[0] + -1894007588, 14) + c; a = RL(a, 10);
	c = RL(c + F4(d, e, a) + X[8] + -1894007588, 15) + b; e = RL(e, 10);
	b = RL(b + F4(c, d, e) + X[12] + -1894007588, 9) + a; d = RL(d, 10);
	a = RL(a + F4(b, c, d) + X[4] + -1894007588, 8) + e; c = RL(c, 10);
	e = RL(e + F4(a, b, c) + X[13] + -1894007588, 9) + d; b = RL(b, 10);
	d = RL(d + F4(e, a, b) + X[3] + -1894007588, 14) + c; a = RL(a, 10);
	c = RL(c + F4(d, e, a) + X[7] + -1894007588, 5) + b; e = RL(e, 10);
	b = RL(b + F4(c, d, e) + X[15] + -1894007588, 6) + a; d = RL(d, 10);
	a = RL(a + F4(b, c, d) + X[14] + -1894007588, 8) + e; c = RL(c, 10);
	e = RL(e + F4(a, b, c) + X[5] + -1894007588, 6) + d; b = RL(b, 10);
	d = RL(d + F4(e, a, b) + X[6] + -1894007588, 5) + c; a = RL(a, 10);
	c = RL(c + F4(d, e, a) + X[2] + -1894007588, 12) + b; e = RL(e, 10);
	cc = RL(cc + F2(dd, ee, aa) + X[8] + 2053994217, 15) + bb; ee = RL(ee, 10);
	bb = RL(bb + F2(cc, dd, ee) + X[6] + 2053994217, 5) + aa; dd = RL(dd, 10);
	aa = RL(aa + F2(bb, cc, dd) + X[4] + 2053994217, 8) + ee; cc = RL(cc, 10);
	ee = RL(ee + F2(aa, bb, cc) + X[1] + 2053994217, 11) + dd; bb = RL(bb, 10);
	dd = RL(dd + F2(ee, aa, bb) + X[3] + 2053994217, 14) + cc; aa = RL(aa, 10);
	cc = RL(cc + F2(dd, ee, aa) + X[11] + 2053994217, 14) + bb; ee = RL(ee, 10);
	bb = RL(bb + F2(cc, dd, ee) + X[15] + 2053994217, 6) + aa; dd = RL(dd, 10);
	aa = RL(aa + F2(bb, cc, dd) + X[0] + 2053994217, 14) + ee; cc = RL(cc, 10);
	ee = RL(ee + F2(aa, bb, cc) + X[5] + 2053994217, 6) + dd; bb = RL(bb, 10);
	dd = RL(dd + F2(ee, aa, bb) + X[12] + 2053994217, 9) + cc; aa = RL(aa, 10);
	cc = RL(cc + F2(dd, ee, aa) + X[2] + 2053994217, 12) + bb; ee = RL(ee, 10);
	bb = RL(bb + F2(cc, dd, ee) + X[13] + 2053994217, 9) + aa; dd = RL(dd, 10);
	aa = RL(aa + F2(bb, cc, dd) + X[9] + 2053994217, 12) + ee; cc = RL(cc, 10);
	ee = RL(ee + F2(aa, bb, cc) + X[7] + 2053994217, 5) + dd; bb = RL(bb, 10);
	dd = RL(dd + F2(ee, aa, bb) + X[10] + 2053994217, 15) + cc; aa = RL(aa, 10);
	cc = RL(cc + F2(dd, ee, aa) + X[14] + 2053994217, 8) + bb; ee = RL(ee, 10);
	b = RL(b + F5(c, d, e) + X[4] + -1454113458, 9) + a; d = RL(d, 10);
	a = RL(a + F5(b, c, d) + X[0] + -1454113458, 15) + e; c = RL(c, 10);
	e = RL(e + F5(a, b, c) + X[5] + -1454113458, 5) + d; b = RL(b, 10);
	d = RL(d + F5(e, a, b) + X[9] + -1454113458, 11) + c; a = RL(a, 10);
	c = RL(c + F5(d, e, a) + X[7] + -1454113458, 6) + b; e = RL(e, 10);
	b = RL(b + F5(c, d, e) + X[12] + -1454113458, 8) + a; d = RL(d, 10);
	a = RL(a + F5(b, c, d) + X[2] + -1454113458, 13) + e; c = RL(c, 10);
	e = RL(e + F5(a, b, c) + X[10] + -1454113458, 12) + d; b = RL(b, 10);
	d = RL(d + F5(e, a, b) + X[14] + -1454113458, 5) + c; a = RL(a, 10);
	c = RL(c + F5(d, e, a) + X[1] + -1454113458, 12) + b; e = RL(e, 10);
	b = RL(b + F5(c, d, e) + X[3] + -1454113458, 13) + a; d = RL(d, 10);
	a = RL(a + F5(b, c, d) + X[8] + -1454113458, 14) + e; c = RL(c, 10);
	e = RL(e + F5(a, b, c) + X[11] + -1454113458, 11) + d; b = RL(b, 10);
	d = RL(d + F5(e, a, b) + X[6] + -1454113458, 8) + c; a = RL(a, 10);
	c = RL(c + F5(d, e, a) + X[15] + -1454113458, 5) + b; e = RL(e, 10);
	b = RL(b + F5(c, d, e) + X[13] + -1454113458, 6) + a; d = RL(d, 10);
	bb = RL(bb + F1(cc, dd, ee) + X[12], 8) + aa; dd = RL(dd, 10);
	aa = RL(aa + F1(bb, cc, dd) + X[15], 5) + ee; cc = RL(cc, 10);
	ee = RL(ee + F1(aa, bb, cc) + X[10], 12) + dd; bb = RL(bb, 10);
	dd = RL(dd + F1(ee, aa, bb) + X[4], 9) + cc; aa = RL(aa, 10);
	cc = RL(cc + F1(dd, ee, aa) + X[1], 12) + bb; ee = RL(ee, 10);
	bb = RL(bb + F1(cc, dd, ee) + X[5], 5) + aa; dd = RL(dd, 10);
	aa = RL(aa + F1(bb, cc, dd) + X[8], 14) + ee; cc = RL(cc, 10);
	ee = RL(ee + F1(aa, bb, cc) + X[7], 6) + dd; bb = RL(bb, 10);
	dd = RL(dd + F1(ee, aa, bb) + X[6], 8) + cc; aa = RL(aa, 10);
	cc = RL(cc + F1(dd, ee, aa) + X[2], 13) + bb; ee = RL(ee, 10);
	bb = RL(bb + F1(cc, dd, ee) + X[13], 6) + aa; dd = RL(dd, 10);
	aa = RL(aa + F1(bb, cc, dd) + X[14], 5) + ee; cc = RL(cc, 10);
	ee = RL(ee + F1(aa, bb, cc) + X[0], 15) + dd; bb = RL(bb, 10);
	dd = RL(dd + F1(ee, aa, bb) + X[3], 13) + cc; aa = RL(aa, 10);
	cc = RL(cc + F1(dd, ee, aa) + X[9], 11) + bb; ee = RL(ee, 10);
	bb = RL(bb + F1(cc, dd, ee) + X[11], 11) + aa; dd = RL(dd, 10);
	dd += c + H[1];

	H[1] = H[2] + d + ee;
	H[2] = H[3] + e + aa;
	H[3] = H[4] + a + bb;
	H[4] = H[0] + b + cc;
	H[0] = dd;

	for (int a11 = 0; a11 != XLen0; a11++)
	{
		X[a11] = 0;
	}

	return H;
}

int main(void)
{
	printf("Copyright (C) 2014 Sean Bradley\n\n");
	printf("Permission is hereby granted, free of charge, to any person obtaining a copy of this software and\nassociated documentation files (the 'Software'),\nto deal in the Software without restriction,\nincluding without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense,\nand/or sell copies of the Software,\nand to permit persons to whom the Software is furnished to do so, subject to the following conditions:\n");
	printf("The above copyright notice and this permission notice shall be included in all copies or substantialnportions of the Software.\n");
	printf("\nTHE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT\nLIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.\nIN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,\nWHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THEnSOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.\n");
	printf("\nSean Bradley\n\n");

	int N = 65536;
	int block_size = 256;
	int n_blocks = N / block_size + (N%block_size == 0 ? 0 : 1);

	printf("%d %d\n", block_size, n_blocks);

	int host_responseLength, dev_responseLength;
	host_responseLength = 20;

	unsigned char *host_response, *dev_response;
	size_t size = 20 * sizeof(unsigned char);
	host_response = (unsigned char *)malloc(size);
	for (int i = 0; i < size; i++)
	{
		host_response[i] = 0;
	}
	hipMalloc((void **)&dev_response, size);
	hipMemcpy(dev_response, host_response, size, hipMemcpyHostToDevice);

	printf("response buffer size=%d\n", size);
	//password = 2c 08 e8 f5 88 47 50 a7 b9 9f 6f 2f 34 2f c6 38 db 25 ff 31
	//the RIPEMD160 to find
	unsigned char *host_query, *dev_query;
	size = 20 * sizeof(unsigned char);
	host_query = (unsigned char *)malloc(size);
	host_query[0] = 0x2C;
	host_query[1] = 0x08;
	host_query[2] = 0xE8;
	host_query[3] = 0xF5;
	host_query[4] = 0x88;
	host_query[5] = 0x47;
	host_query[6] = 0x50;
	host_query[7] = 0xA7;
	host_query[8] = 0xB9;
	host_query[9] = 0x9F;
	host_query[10] = 0x6F;
	host_query[11] = 0x2F;
	host_query[12] = 0x34;
	host_query[13] = 0x2F;
	host_query[14] = 0xC6;
	host_query[15] = 0x38;
	host_query[16] = 0xDB;
	host_query[17] = 0x25;
	host_query[18] = 0xFF;
	host_query[19] = 0x31;
	hipMalloc((void **)&dev_query, size);
	hipMemcpy(dev_query, host_query, size, hipMemcpyHostToDevice);

	std::clock_t start;

	start = std::clock();
	kernel <<<n_blocks, block_size >>> (dev_response, 20, dev_query, 20, N);
	hipMemcpy(host_response, dev_response, sizeof(unsigned char) * 20, hipMemcpyDeviceToHost);
	printf("ms = %lu.nn", (std::clock() - start));

	hipFree(dev_response); hipFree(dev_query);

	for (int j = 0; j < host_responseLength; j++)
	{
		//printf("%02X", host_response[j + (i * 20)]);
		printf("%02X", host_response[j]);
	}
	printf("n");

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%sn", hipGetErrorString(err));

	free(host_response);

	printf("\nCopyright Sean Bradley 2014\n\n");

	system("pause");

}
